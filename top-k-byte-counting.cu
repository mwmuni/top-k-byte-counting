#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include <algorithm>

#define BINS 256
#define NUM_BYTES sizeof(int)
#define K 3

// CUDA error checking macro
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
    if (code != hipSuccess){
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Kernel to count byte frequencies
__global__ void count_byte_frequencies(const int *data, int n, int byte_pos, int *histogram) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int byte_value = (data[idx] >> (8 * (NUM_BYTES - 1 - byte_pos))) & 0xFF;
        atomicAdd(&histogram[byte_value], 1);
    }
}

// Kernel to filter data based on byte value
__global__ void filter_data(const int *data, int n, int byte_pos, int byte_value, int *filtered_data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int current_byte = (data[idx] >> (8 * (NUM_BYTES - 1 - byte_pos))) & 0xFF;
        if (current_byte == byte_value) {
            // Use atomic to find the index to place the element
            int pos = atomicAdd(&filtered_data[n], 1);
            filtered_data[pos] = data[idx];
        }
    }
}

// Host function to find top K using CUDA
void find_top_k_cuda(const std::vector<int>& host_data, std::vector<int>& top_k) {
    int n = host_data.size();
    int *d_data, *d_histogram, *d_filtered_data;

    // Allocate device memory
    cudaCheckError(hipMalloc(&d_data, n * sizeof(int)));
    cudaCheckError(hipMalloc(&d_histogram, BINS * sizeof(int)));
    cudaCheckError(hipMalloc(&d_filtered_data, (n + 1) * sizeof(int))); // Extra space for count

    // Initialize filtered_data count to 0
    cudaCheckError(hipMemset(d_filtered_data, 0, (n + 1) * sizeof(int))); // Corrected size

    // Copy data to device
    cudaCheckError(hipMemcpy(d_data, host_data.data(), n * sizeof(int), hipMemcpyHostToDevice));

    int byte_pos = 0;
    std::vector<int> current_data = host_data;
    int current_k = 0;

    while (byte_pos < NUM_BYTES && current_k < K) {
        n = current_data.size();

        // Reset histogram
        cudaCheckError(hipMemset(d_histogram, 0, BINS * sizeof(int)));

        // Copy current data to device
        cudaCheckError(hipMemcpy(d_data, current_data.data(), n * sizeof(int), hipMemcpyHostToDevice));

        // Launch kernel to count byte frequencies
        int threads = 256;
        int blocks = (n + threads - 1) / threads;
        count_byte_frequencies<<<blocks, threads>>>(d_data, n, byte_pos, d_histogram);
        cudaCheckError(hipDeviceSynchronize());

        // Copy histogram back to host
        std::vector<int> host_histogram(BINS, 0);
        cudaCheckError(hipMemcpy(host_histogram.data(), d_histogram, BINS * sizeof(int), hipMemcpyDeviceToHost));

        // Iterate over histogram from high to low
        for (int i = BINS - 1; i >= 0 && current_k < K; --i) {
            if (host_histogram[i] == 0) continue;

            if (current_k + host_histogram[i] <= K) {
                // Launch kernel to add all numbers with byte_value == i to top_k
                // First, reset filtered_data count to 0
                cudaCheckError(hipMemset(d_filtered_data, 0, (n + 1) * sizeof(int))); // Corrected size

                // Launch filter kernel
                filter_data<<<blocks, threads>>>(d_data, n, byte_pos, i, d_filtered_data);
                cudaCheckError(hipDeviceSynchronize());

                // Copy filtered data count
                int count;
                cudaCheckError(hipMemcpy(&count, d_filtered_data + n, sizeof(int), hipMemcpyDeviceToHost));

                if (count > 0) {
                    std::vector<int> temp(count);
                    // Check if count is valid
                    if (count < 0 || count > n) {
                        std::cerr << "Invalid count: " << count << std::endl;
                        return; // or handle the error appropriately
                    }

                    // Perform the memcpy
                    cudaCheckError(hipMemcpy(temp.data(), d_filtered_data, count * sizeof(int), hipMemcpyDeviceToHost));

                    for (int num : temp) {
                        top_k.push_back(num);
                        current_k++;
                        if (current_k >= K) break;
                    }
                }
            } else {
                // Need to further partition this bin by the next byte
                // Launch filter kernel to get numbers with byte_value == i
                cudaCheckError(hipMemset(d_filtered_data, 0, (n + 1) * sizeof(int))); // Corrected size

                filter_data<<<blocks, threads>>>(d_data, n, byte_pos, i, d_filtered_data);
                cudaCheckError(hipDeviceSynchronize());

                // Copy filtered data count
                int count;
                cudaCheckError(hipMemcpy(&count, d_filtered_data + n, sizeof(int), hipMemcpyDeviceToHost));

                if (count > 0) {
                    std::vector<int> temp(count);
                    // Check if count is valid
                    if (count < 0 || count > n) {
                        std::cerr << "Invalid count: " << count << std::endl;
                        return; // or handle the error appropriately
                    }

                    // Perform the memcpy
                    cudaCheckError(hipMemcpy(temp.data(), d_filtered_data, count * sizeof(int), hipMemcpyDeviceToHost));
                    current_data = temp;
                }
                break; // Proceed to next byte_pos
            }
        }

        byte_pos++;
    }

    // Free device memory
    hipFree(d_data);
    hipFree(d_histogram);
    hipFree(d_filtered_data);
}

int main() {
    std::vector<int> data = {399, 18, 512, 1024, 42, 123, 34, 255, 67, 89,
                             101, 44, 111, 222, 333, 444, 555, 666, 777, 888, 999};
    std::vector<int> top_k;
    find_top_k_cuda(data, top_k);

    std::cout << "Top " << K << " numbers are:\n";
    for (int num : top_k) {
        std::cout << num << '\n';
    }

    return 0;
}